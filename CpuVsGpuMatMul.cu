#include <hip/hip_runtime.h>
#include <nvtx3/nvToolsExt.h>
#include <iostream>


// #1      #2
//  1 3    2 0
//  4 6    5 4

int[][] cpuMatMul( int &matrix1colCount, int &matrix1rowCount, int[][] &matrix1, int &matrix2colCount, int &matrix2rowCount, int[][] &matrix2) {
    int returnArr[matrix1colCount][matrix2rowCount];
    // 
    for (int i = 0; i < matrix1RowCount){
        int sum = 0;
        // 
        for (int j = 0; j < matrix2rowCount; j++ ){
            returnArr[i][j] = 0;
            // goes through matrix 2 columns
            for (int k = 0; k < matrix2RowCount; k++) {
                returnArr[i][j] = matrix1[i][k]*matrix2[k][j];
            }
        }       
    }

    return returnArr;

}

int main() {
    int matLen = 2
    int[matLen][matLen] matrix1 = {{1,3},{4,6}};
    int[matLen][matLen] matrix2 = {{2,0},{5,4}};
    int[matLen][matLen] cpuMatMulOutput = cpuMatMul(2,2, matrix1, 2,2, matrix2);
    
  
  return 0;
}