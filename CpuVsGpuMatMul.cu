#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <nvtx3/nvToolsExt.h>
#include <iostream>
using namespace std;



// #1      #2
//  1 3    2 0
//  4 6    5 4

// 15
// 12
// 30
// 24
void cpuMatMul( int matrix1ColCount, int matrix1RowCount, auto  &matrix1, int matrix2ColCount, int matrix2RowCount, auto &matrix2) {
    int returnArr[matrix1ColCount][matrix2RowCount];
    // 
    for (int i = 0; i < matrix1RowCount; i++){
        int sum = 0;
        // 
        for (int j = 0; j < matrix2RowCount; j++ ){
            returnArr[i][j] = 0;
            // goes through matrix 2 columns
            for (int k = 0; k < matrix2RowCount; k++) {
                returnArr[i][j] += matrix1[i][k]*matrix2[k][j];
                
            }
            cout << returnArr[i][j] << endl;
        }       
    }

    // return returnArr;

}

__global__ void gpuMatMul(float A[2][2], float B[2][2], float C[2][2])
{
	int i = threadIdx.x;
	int j = threadIdx.y;
	C[i][j] = A[i][j] + B[i][j];
	// cout << C[i][j] << endl;
}
	

int main() {
    int matLen = 2;
    int matrix1[2][2] = {{1,3},{4,6}};
    int matrix2[2][2] = {{2,0},{5,4}};
    cpuMatMul(2,2, matrix1, 2,2, matrix2);
	
    
    int A[2][2] = {{2,0},{5,4}};
    int B[2][2] = {{1,3},{4,6}};
    int C[2][2];
	dim3 threadsPerBlock(2, 2);
	gpuMatMul<<<1, threadsPerBlock>>>(A, B, C);
    
  
  return 0;
}

