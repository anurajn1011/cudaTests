#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <nvtx3/nvToolsExt.h>
#include <iostream>
#include <random>

#define N 1000

using namespace std;


// #1      #2
//  1 3    2 0
//  4 6    5 4

// 15
// 12
// 30
// 24
void cpuMatMul( int matrix1ColCount, int matrix1RowCount, auto  &matrix1, int matrix2ColCount, int matrix2RowCount, auto &matrix2) {
    int returnArr[matrix1ColCount][matrix2RowCount];
    // 
    for (int i = 0; i < matrix1RowCount; i++){
        int sum = 0;
        // 
        for (int j = 0; j < matrix2RowCount; j++ ){
            returnArr[i][j] = 0;
            // goes through matrix 2 columns
            for (int k = 0; k < matrix2RowCount; k++) {
                returnArr[i][j] += matrix1[i][k]*matrix2[k][j];
                
            }
            cout << returnArr[i][j] << endl;
        }       
    }

    // return returnArr;

}

// ordinary multipliction, not GMEM
__global__ void gpuMatMul(int M, int N, int K, int &matrix1, int &matrix2, int &res)
{
	// the thread locates the indices of C for which it is responsible for
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (i < M && j < N) {
		for (int i = 0; i < K; ++i) {
			res[x * N + y] += matrix1[x * K + i] * matrix2[i * N + y];
		}
	}
}
	

int main() {
    // int matLen = 2;
    //int matrix1[2][2] = {{1,3},{4,6}};
    //int matrix2[2][2] = {{2,0},{5,4}};
    //cpuMatMul(2,2, matrix1, 2,2, matrix2);
	
    
   // int A[2][2] = {{2,0},{5,4}};
    //int B[2][2] = {{1,3},{4,6}};
    //int C[2][2];
	//dim3 threadsPerBlock(2, 2);
	//gpuMatMul<<<1, threadsPerBlock>>>(A, B, C);
	
	int **matrix1, **matrix2, **res;
	int **deviceMatrix1, **deviceMatrix2, **deviceRes;
	size_t size = N * N * sizeof(int); // N is a macro
	
	// allocate host memory
	matrix1 = (*int)malloc(size);
	matrix2 = (*int)malloc(size);
	res = (*int)malloc(size);
	
	// creating host arrays, 
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < N; ++j) {
			matrix1[i][j] = j;
			matrix2[i][j] = j;
			res[i][j] = 0;
		}
	}
	
	// allocation of memory
	hipMalloc(&deviceMatrix1, size);
	hipMalloc(&deviceMatrix2, size);
	hipMalloc(&deviceRes, size);
	
	// move data from host to device
	hipMemcpy(deviceMatrix1, matrix1, size, hipMemcpyHostToDevice);
	hipMemcpy(deviceMatrix2, matrix2, size, hipMemcpyHostToDevice);
	
	// run kernel
	dim3 gridDim(CEIL_DIV(N, 32), CEIL_DIV(N, 32), 1);
	dim3 blockDim(32, 32, 1);
	gpuMatMul<<<gridDim, blockDim>>>(N, N, N, deviceMatrix1, deviceMatrix2, deviceRes);
	
	// moving result to host
	hipMemcpy(res, deviceRes, hipMemcpyDeviceToHost);
	
	// freeing memory
	free(matrix1);
	free(matrix2)
	free(res)
	
	hipFree(deviceMatrix1);
	hipFree(deviceMatrix2);
	hipFree(deviceRes);
    
  
  return 0;
}

