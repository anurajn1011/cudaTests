#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <nvtx3/nvToolsExt.h>
#include <iostream>
#include <random>
#include <cmath>
#include <stdio.h>


#define MACRO_N 3

using namespace std;


// #1      #2
//  1 3    2 0
//  4 6    5 4

// 15
// 12
// 30
// 24
void cpuMatMul( int matrix1ColCount, int matrix1RowCount, auto  &matrix1, int matrix2ColCount, int matrix2RowCount, auto &matrix2) {
    int returnArr[matrix1ColCount][matrix2RowCount];
    // 
    for (int i = 0; i < matrix1RowCount; i++){
        int sum = 0;
        // 
        for (int j = 0; j < matrix2RowCount; j++ ){
            returnArr[i][j] = 0;
            // goes through matrix 2 columns
            for (int k = 0; k < matrix2RowCount; k++) {
                returnArr[i][j] += matrix1[i][k]*matrix2[k][j];
                
            }
            cout << returnArr[i][j] << endl;
        }       
    }

    // return returnArr;

}

// ordinary multipliction, not GMEM
__global__ void gpuMatMul(int M, int N, int K, int *matrix1, int *matrix2, int *res)
{
	// the thread locates the indices of C for which it is responsible for
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (x < M && y < N) {
		int sum = 0;
		for (int i = 0; i < K; ++i) {
			sum += matrix1[x * K + i] * matrix2[i * N + y];
			
            /*# if __CUDA_ARCH__>=200
            printf("%$d", res[x * N + y]);
            #endif*/
		}
		res[x * N + y] = sum; // explicit definition of indices, GPU isn't immediately zero'd.
	}
}
	

int main() {
    cout<< "57" <<endl;
    // int matLen = 2;
    //int matrix1[2][2] = {{1,3},{4,6}};
    //int matrix2[2][2] = {{2,0},{5,4}};
    //cpuMatMul(2,2, matrix1, 2,2, matrix2);
	
    
   // int A[2][2] = {{2,0},{5,4}};
    //int B[2][2] = {{1,3},{4,6}};
    //int C[2][2];
	//dim3 threadsPerBlock(2, 2);
	//gpuMatMul<<<1, threadsPerBlock>>>(A, B, C);
	
	int *matrix1, *matrix2, *res;
	int *deviceMatrix1, *deviceMatrix2, *deviceRes;
	size_t size = MACRO_N * MACRO_N * sizeof(int); // N is a macro
	
    cout<< "74" <<endl;
	// allocate host memory
	matrix1 = (int*)malloc(size);
	matrix2 = (int*)malloc(size);
	res = (int*)malloc(size);

    cout<< "80" <<endl;
	
	// creating host arrays, 
	for (int i = 1; i < MACRO_N*MACRO_N + 1; ++i) {
		matrix1[i-1] = i;
		matrix2[i-1] = i;
	}
	

	for (int i = 0; i <MACRO_N*MACRO_N; i++){
        cout<< matrix1[i]  << " " ;
    }

    cout<< "80" <<endl;
	// allocation of memory
	hipMalloc(&deviceMatrix1, size);
	hipMalloc(&deviceMatrix2, size);
	hipMalloc(&deviceRes, size);
	
    cout<< "95" <<endl;

	// move data from host to device
	hipMemcpy(deviceMatrix1, matrix1, size, hipMemcpyHostToDevice);
	hipMemcpy(deviceMatrix2, matrix2, size, hipMemcpyHostToDevice);
	// hipMemcpy(deviceRes, res, size, hipMemcpyHostToDevice); only do this if res has initial vals

    cout<< "101" <<endl;

	// run kernel
	dim3 threadsPerBlock(MACRO_N, MACRO_N);
    dim3 blocksPerGrid(1, 1);
	gpuMatMul<<<blocksPerGrid, threadsPerBlock>>>(MACRO_N, MACRO_N, MACRO_N, deviceMatrix1, deviceMatrix2, deviceRes);
	
    cout<< "108" <<endl;

	// moving result to host
	hipMemcpy(res, deviceRes, size, hipMemcpyDeviceToHost);
	
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		cout << "CUDA Error: " << hipGetErrorString(error) << endl;
	}

    cout<< "113" <<endl;


    cout<< "119" <<endl;

    for (int i = 0; i <MACRO_N*MACRO_N; i++){
        cout<< res[i]  << " " ;
    }
    cout<< endl;
	
    cout<< "126" <<endl;
	// freeing memory
	free(matrix1);
	free(matrix2);
	free(res);
	
	hipFree(deviceMatrix1);
	hipFree(deviceMatrix2);
	hipFree(deviceRes);
    
  
  return 0;
}








// 15 18 21 
// 42 54 66
// 69 90 111