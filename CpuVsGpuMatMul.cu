#include <hip/hip_runtime.h>
#include <nvtx3/nvToolsExt.h>
#include <iostream>

5 3 7
8 2 9

int[] cpuMatMul( int matrix1colCount, int matrix1rowCount, int[] matrix1, int matrix2colCount, int matrix2rowCount, int[] matrix2) {
    int returnArr[matrix2colCount];
    // for loop of column count of matrix 2 
    for (int i = 0; i < matrix2columnCount){
        int sum  = 0
        // for loop on column count of matrix 1
        for (int j = 0; j < matrix1colCount; j++ ){
            int product = matrix1[j]*matrix2[j*matrix2colCount];
            sum += product;
        }
        returnArr[i] = sum;
        
    }

    return returnArr


}

int main() {
  
  return 0;
}