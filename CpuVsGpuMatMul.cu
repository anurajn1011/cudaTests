#include <hip/hip_runtime.h>
#include <nvtx3/nvToolsExt.h>
#include <iostream>
using namespace std;



// #1      #2
//  1 3    2 0
//  4 6    5 4

// 15
// 12
// 30
// 24
void cpuMatMul( int matrix1ColCount, int matrix1RowCount, auto  &matrix1, int matrix2ColCount, int matrix2RowCount, auto &matrix2) {
    int returnArr[matrix1ColCount][matrix2RowCount];
    // 
    for (int i = 0; i < matrix1RowCount; i++){
        int sum = 0;
        // 
        for (int j = 0; j < matrix2RowCount; j++ ){
            returnArr[i][j] = 0;
            // goes through matrix 2 columns
            for (int k = 0; k < matrix2RowCount; k++) {
                returnArr[i][j] += matrix1[i][k]*matrix2[k][j];
                
            }
            cout << returnArr[i][j] << endl;
        }       
    }

    // return returnArr;

}

int main() {
    int matLen = 2;
    int matrix1[2][2] = {{1,3},{4,6}};
    int matrix2[2][2] = {{2,0},{5,4}};
    cpuMatMul(2,2, matrix1, 2,2, matrix2);
    
  
  return 0;
}

